// include any headers

#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// define constants
#define BLOCK_SIZE 256
#define SIZE 100

// function to apply scan on arrays
__global__ void scan(int *in, int *out, int n){
    __shared__ int temp[BLOCK_SIZE], source[BLOCK_SIZE], dest[BLOCK_SIZE];
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = 1;

    // Compute
    if (idx < n) {
        temp[threadIdx.x] = in[idx];
        source[threadIdx.x] = in[idx];
        dest[threadIdx.x] = out[idx];
    }
    __syncthreads();

     for (int i = 1; i < blockDim.x; i++) {
        //int tempT;
        if (threadIdx.x >= i) {
            dest[threadIdx.x] = source[i] + source[i-stride];
            temp[i] = dest[threadIdx.x];
            dest[i] = source[i];
            source[i] = temp[i];
            stride = stride *2;
        }
        __syncthreads();
        //if (threadIdx.x >= i) {
        //    temp[threadIdx.x] += tempT;
        //}
        //__syncthreads();
    }
    if(idx < n){
        out[idx] = dest[threadIdx.x];
    }
}

int main(void) {
  // allocate input and output arrays
    int *input, *output;
    hipMallocManaged(&input, SIZE*sizeof(int));
    hipMallocManaged(&output, SIZE*sizeof(int));
  // initialize input array on the host
    for(int i=0; i<SIZE; i++){
        input[i] = 1;
    }

  // run the kernel
    scan<<<(SIZE + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(input, output, SIZE);
  // check for errors
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(error) << std::endl;
    } else {
        std::cout << "Kernel completed successfully!" << std::endl;
    }

    // print out input and output array
    std::cout << "Input array:" << std::endl;
    for (int i = 0; i < SIZE; i++){
    std::cout << input[i] << " ";
    }
    std::cout << std::endl;

    std::cout << "Output array:" << std::endl;
    for (int i = 0; i < SIZE; i++) {
        std::cout << output[i] << " ";
    }
    std::cout << std::endl;

  // free memory
    hipFree(input);
    hipFree(output);
  return 0;
}
