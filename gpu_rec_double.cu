// include any headers

#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// define constants
#define BLOCK_SIZE 256
#define SIZE 10000

// function to apply scan on arrays
__global__ void scan(int *in, int *out, int n){
    __shared__ int tempOne[BLOCK_SIZE];
    __shared__ int tempTwo[BLOCK_SIZE];
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Load input into shared memory
    if (idx < n) {
        tempOne[threadIdx.x] = in[idx];
    }
    __syncthreads();

    int* source = tempOne;
    int* dest = tempTwo;

     // Koggle-stone calculation
     for (int i = 1; i < blockDim.x; i*=2) {
        if (threadIdx.x >= i) {
            dest[threadIdx.x] = source[threadIdx.x] + source[threadIdx.x-i];
        }
        else{
            dest[threadIdx.x] = source[threadIdx.x];
        }

        // swipe source and dest
        int* temp = source;
        source = dest;
        dest = temp;
    }

    // Put results back to global memory
    if(idx < n){
        out[idx] = dest[threadIdx.x];
    }
}

int main(void) {
  // allocate input and output arrays
    int *input, *output;
    hipMallocManaged(&input, SIZE*sizeof(int));
    hipMallocManaged(&output, SIZE*sizeof(int));
  // initialize input array on the host
    for(int i=0; i<SIZE; i++){
        input[i] = 1;
    }

  // run the kernel
    scan<<<(SIZE + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(input, output, SIZE);
  // check for errors
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(error) << std::endl;
    } else {
        std::cout << "Kernel completed successfully!" << std::endl;
    }

    // print out input and output array
    //std::cout << "Input array:" << std::endl;
    //for (int i = 0; i < SIZE; i++){
    //std::cout << input[i] << " ";
    //}
    //std::cout << std::endl;

    //std::cout << "Output array:" << std::endl;
    //for (int i = 0; i < SIZE; i++) {
    //    std::cout << output[i] << " ";
    //}
    //std::cout << std::endl;

  // free memory
    hipFree(input);
    hipFree(output);
  return 0;
}
