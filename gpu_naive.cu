// include any headers

#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// define constants
#define BLOCK_SIZE 256
#define SIZE 100000

// function to apply scan on arrays
__global__ void scan(int *in, int *out, int n){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Compute
    if (idx < n) {
        int sum = 0;
        for (int i = 0; i <= idx; i++) {
            // Add all in[i] elements
            sum += in[i];
        }
        out[idx] = sum;
    }
}

int main(void) {
  // allocate input and output arrays
    int *input, *output;
    hipMallocManaged(&input, SIZE*sizeof(int));
    hipMallocManaged(&output, SIZE*sizeof(int));
  // initialize input array on the host
    for(int i=0; i<SIZE; i++){
        input[i] = 1;
    }

  // run the kernel
    scan<<<(SIZE + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(input, output, SIZE);
  // check for errors
    hipDeviceSynchronize();

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(error) << std::endl;
    } else {
        std::cout << "Kernel completed successfully!" << std::endl;
    }

    // print out input and output array
    //std::cout << "Input array:" << std::endl;
    //for (int i = 0; i < SIZE; i++)
    //std::cout << input[i] << " ";
    //}
    //std::cout << std::endl;

    //std::cout << "Output array:" << std::endl;
    //for (int i = 0; i < SIZE; i++) {
    //    std::cout << output[i] << " ";
    //}
    //std::cout << std::endl;

  // free memory
    hipFree(input);
    hipFree(output);
  return 0;
}
